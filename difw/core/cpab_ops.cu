#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <float.h>

// FUNCTIONS

#define eps FLT_EPSILON;
#define inf INFINITY;

__device__ int sign(const int r){
    return (r > 0) - (r < 0);
}

__device__ int signf(const float r){
    return (r > 0) - (r < 0);
}

__device__ bool cmpf(float x, float y){
    return fabs(x - y) < eps;
}

__device__ bool cmpf0(const float& x){
    return fabs(x) < eps;
}

__device__ float right_boundary(const int& c, const float& xmin, const float& xmax, const int& nc){
    return xmin + (c + 1) * (xmax - xmin) / nc + eps;
}

__device__ float left_boundary(const int& c, const float& xmin, const float& xmax, const int& nc){
    return xmin + c * (xmax - xmin) / nc - eps;
}

__device__ int get_cell(const float& x, const float& xmin, const float& xmax, const int& nc){
    int c = floor((x - xmin) / (xmax - xmin) * nc);
    c = max(0, min(c, nc-1));
    return c;
}

__device__ float get_velocity(const float& x, const float* A, const int& n_batch, const int& batch_index, const float& xmin, const float& xmax, const int& nc){
    const int c = get_cell(x, xmin, xmax, nc);
    const float a = A[(2*c) * n_batch + batch_index];
    const float b = A[(2*c+1) * n_batch + batch_index];
    return a*x + b;
}

__device__ float get_velocity_dx(const float& x, const float* A, const int& n_batch, const int& batch_index, const float& xmin, const float& xmax, const int& nc){
    const int c = get_cell(x, xmin, xmax, nc);
    const float a = A[(2*c) * n_batch + batch_index];
    return a;
}

// INTEGRATION CLOSED FORM

__device__ float get_psi(const float& x, const float& t,  const float& a, const float& b){
    if (cmpf0(a)){
        return x + t*b;
    }
    else{
        return exp(t*a) * (x + (b/a)) - (b/a);
    }
}

__device__ float get_hit_time(float x, float t, int c, const float& a, const float& b, const float& xmin, const float& xmax, const int& nc, float& xc, int& cc){
    int s = signf(t);
    float infinite = inf*s;

    const float v = a * x + b;
    if(cmpf0(v)) return infinite;

    cc = c + signf(v)*signf(t);
    if(cc < 0 || cc >= nc) return infinite;
    // xc = (v > 0) ? right_boundary(c, xmin, xmax, nc) : left_boundary(c, xmin, xmax, nc);
    if (t > 0){
        xc = (v > 0) ? right_boundary(c, xmin, xmax, nc) : left_boundary(c, xmin, xmax, nc);
    }else{
        xc = (v > 0) ? left_boundary(c, xmin, xmax, nc) : right_boundary(c, xmin, xmax, nc);
    }

    const float vc = a * xc + b;
    if(cmpf0(vc)) return infinite;
    if(signf(v) != signf(vc)) return infinite;
    if(xc == xmin || xc == xmax) return infinite;

    if(cmpf0(a)){
        return (xc - x)/b;
    }else{
        return std::log(vc / v) / a;
    }
}

__device__ float integrate_closed_form(float x, float t, const float* A, const int& n_batch, const int& batch_index, const float& xmin, const float& xmax, const int& nc){
    int c = get_cell(x, xmin, xmax, nc);
    int cont = 0;
    const int contmax = std::max(c, nc-1-c);
    int s = signf(t);

    float a, b, xc, thit;
    int cc;
    while (true) {
        a = A[(2*c) * n_batch + batch_index];
        b = A[(2*c+1) * n_batch + batch_index];

        thit = get_hit_time(x, t, c, a, b, xmin, xmax, nc, xc, cc);
        if (s*thit > s*t){
            return get_psi(x, t, a, b);
        }

        x = xc;
        c = cc;
        t -= thit;

        cont++;
        if (cont > contmax){
            break;
        }
    }
    return -1;
}

__device__ void integrate_closed_form_trace(float* result, float x, float t, const float* A, const int& n_batch, const int& batch_index, const float& xmin, const float& xmax, const int& nc){
    int c = get_cell(x, xmin, xmax, nc);
    int cont = 0;
    const int contmax = std::max(c, nc-1-c);
    int s = signf(t);

    float a, b, xc, thit;
    int cc;
    while (true) {
        a = A[(2*c) * n_batch + batch_index];
        b = A[(2*c+1) * n_batch + batch_index];

        thit = get_hit_time(x, t, c, a, b, xmin, xmax, nc, xc, cc);
        if (s*thit > s*t){
            result[0] = get_psi(x, t, a, b);
            result[1] = t;
            result[2] = c;
            return;
        }

        x = xc;
        c = cc;
        t -= thit;

        cont++;
        if (cont > contmax){
            break;
        }
    }
    return;
}

// INTEGRATION NUMERIC

__device__ float get_psi_numeric(const float& x, const int& c, const float& t, const float* A, const int& n_batch, const int& batch_index){
    // const int c = get_cell(x, xmin, xmax, nc);
    const float a = A[(2*c) * n_batch + batch_index];
    const float b = A[(2*c+1) * n_batch + batch_index];
    if (cmpf0(a)){
        return x + t*b;
    }
    else{
        return std::exp(t*a) * (x + (b/a)) - (b/a);
    }
}

__device__ float get_phi_numeric(const float& x, const float& t, const int& nSteps2, const float* A, const int& n_batch, const int& batch_index, const float& xmin, const float& xmax, const int& nc){
    float yn = x;
    float midpoint;
    const float deltaT = t / nSteps2;
    for(int j = 0; j < nSteps2; j++) {
        midpoint = yn + deltaT / 2 * get_velocity(yn, A, n_batch, batch_index, xmin, xmax, nc);
        yn = yn + deltaT * get_velocity(midpoint, A, n_batch, batch_index, xmin, xmax, nc);
    }
    return yn;
}

__device__ float integrate_numeric(const float& x, const float& t, const float* A, const int& n_batch, const int& batch_index, const float& xmin, const float& xmax, const int& nc, const int& nSteps1, const int& nSteps2){
    float xPrev = x;
    const float deltaT = t / nSteps1;
    int c = get_cell(x, xmin, xmax, nc);
    for(int j = 0; j < nSteps1; j++) {
        float xTemp = get_psi_numeric(xPrev, c, deltaT, A, n_batch, batch_index);
        int cTemp = get_cell(xTemp, xmin, xmax, nc);
        if (c == cTemp){
            xPrev = xTemp;
        }
        else{
            xPrev = get_phi_numeric(xPrev, deltaT, nSteps2, A, n_batch, batch_index, xmin, xmax, nc);
            c = get_cell(xPrev, xmin, xmax, nc);
        }
    }
    return xPrev;
}


// DERIVATIVE

__device__ void derivative_psi_theta(double* gradpoints, const float& x, const int& c, const float& t, const int& d, const float* B, const float* A, const int& n_batch, const int& batch_index, const int& n_points, const int& point_index){
    const double a = A[(2*c) * n_batch + batch_index];
    const double b = A[(2*c+1) * n_batch + batch_index];

    
    if (cmpf0(a)){
        for(int k=0; k < d; k++){
            const double ak = B[(2*c)*d + k];
            const double bk = B[(2*c+1)*d + k];
            gradpoints[batch_index*(n_points * d) + point_index*d + k] += t*(x*ak + bk);
        }
    }
    else{
        const double tmp = exp(t*a);
        const double tmp1 = t * tmp * (x + b/a);
        const double tmp2 = (tmp-1)/pow(a, 2.0);
        for(int k=0; k < d; k++){
            const double ak = B[(2*c)*d + k];
            const double bk = B[(2*c+1)*d + k];
            gradpoints[batch_index*(n_points * d) + point_index*d + k] += ak * tmp1 + tmp2 * (bk*a - ak*b);
        }
    }
}

__device__ float derivative_phi_time(const float& x, const int& c, const float& t, const float* A, const int& n_batch, const int& batch_index){
    const double a = A[(2*c) * n_batch + batch_index];
    const double b = A[(2*c+1) * n_batch + batch_index];

    if (cmpf0(a)){
        return b;
    }
    else{
        return exp(t*a)*(a*x + b);
    }
}

__device__ void derivative_thit_theta(double* gradpoints, const float& x, const int& c, const float& xc, const int& d, const float* B, const float* A, const int& n_batch, const int& batch_index, const int& n_points, const int& point_index){
    const double a = A[(2*c) * n_batch + batch_index];
    const double b = A[(2*c+1) * n_batch + batch_index];

    if (cmpf0(a)){
        const double tmp = (x-xc) / pow(b, 2.0);
        for(int k=0; k < d; k++){
            const double bk = B[(2*c+1)*d + k];
            gradpoints[batch_index*(n_points * d) + point_index*d + k] -= tmp*bk;
        }
    }
    else{
        const double tmp1 = log( (a*xc + b) / (a*x + b) )/pow(a, 2.0);
        const double tmp2 = (x - xc) / (a * (a*x + b) * (a*xc + b) );
        for(int k=0; k < d; k++){
            const double ak = B[(2*c)*d + k];
            const double bk = B[(2*c+1)*d + k];

            const double d1 = - ak * tmp1;
            const double d2 = ( bk*a - ak*b) * tmp2;;
            gradpoints[batch_index*(n_points * d) + point_index*d + k] -= d1 + d2;
        }
    }
}

__device__ void derivative_phi_theta(double* gradpoints, const float& xini, const float& tm, const int& cm, const int& d, const float* B, const float* A, const int& n_batch, const int& batch_index, const int& n_points, const int& point_index, const float& xmin, const float& xmax, const int& nc){
    
    const int cini = get_cell(xini, xmin, xmax, nc);
    float xm = xini;

    if (cini != cm){
        float xc;
        const int step = sign(cm - cini);
        for (int c = cini; step*c < cm*step; c += step){
            if (step == 1){
                xc = right_boundary(c, xmin, xmax, nc);
            }else if (step == -1){
                xc = left_boundary(c, xmin, xmax, nc);
            }
            derivative_thit_theta(gradpoints, xm, c, xc, d, B, A, n_batch, batch_index, n_points, point_index);
            xm = xc;
        }
    }

    const float dpsi_dtime = derivative_phi_time(xm, cm, tm, A, n_batch, batch_index);
    for(int k=0; k < d; k++){
        gradpoints[batch_index*(n_points * d) + point_index*d + k] *= dpsi_dtime;
    }
    derivative_psi_theta(gradpoints, xm, cm, tm, d, B, A, n_batch, batch_index, n_points, point_index);
    
}


// KERNELS

__global__ void kernel_get_cell(
    const int n_points, const float* x, 
    const float xmin, const float xmax, const int nc, int* newpoints){  

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    if(point_index < n_points) {
        newpoints[point_index] = get_cell(x[point_index], xmin, xmax, nc);
    }
    return;
}

__global__ void kernel_get_velocity(
    const int n_points, const int n_batch, const float* x, const float* A, 
    const float xmin, const float xmax, const int nc, float* newpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    if(point_index < n_points && batch_index < n_batch) {
        newpoints[batch_index * n_points + point_index] = get_velocity(x[batch_index * n_points + point_index], A, n_batch, batch_index, xmin, xmax, nc);
    }
    return;
}

__global__ void kernel_derivative_velocity_dx(
    const int n_points, const int n_batch, const float* x, const float* A, 
    const float xmin, const float xmax, const int nc, float* newpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    if(point_index < n_points && batch_index < n_batch) {
        newpoints[batch_index * n_points + point_index] = get_velocity_dx(x[batch_index * n_points + point_index], A, n_batch, batch_index, xmin, xmax, nc);
    }
    return;
}

__global__ void kernel_integrate_numeric(
    const int n_points, const int n_batch, const float* x, const float* A, 
    const float t, const float xmin, const float xmax, const int nc, 
    const int nSteps1, const int nSteps2, float* newpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    if(point_index < n_points && batch_index < n_batch) {
        newpoints[batch_index * n_points + point_index] = integrate_numeric(x[batch_index * n_points + point_index], t, A, n_batch, batch_index, xmin, xmax, nc, nSteps1, nSteps2);
    }
    return;
}

__global__ void kernel_integrate_closed_form(
    const int n_points, const int n_batch, const float* x, const float* A, 
    const float t, const float xmin, const float xmax, const int nc, float* newpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    if(point_index < n_points && batch_index < n_batch) {
        newpoints[batch_index * n_points + point_index] = integrate_closed_form(x[batch_index * n_points + point_index], t, A, n_batch, batch_index, xmin, xmax, nc);
    }
    return;
}

__global__ void kernel_derivative_closed_form(
    const int n_points, const int n_batch, const int d,
    const float* x, const float* A, const float* B, 
    const float t, const int xmin, const int xmax, const int nc, double* gradpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    
    const int e = 3;

    if(point_index < n_points && batch_index < n_batch){ 
        float result[e];
        integrate_closed_form_trace(result, x[batch_index * n_points + point_index], t, A, n_batch, batch_index, xmin, xmax, nc);
            
        // float phi = result[0];
        float tm = result[1];
        int cm = result[2];
        derivative_phi_theta(gradpoints, x[batch_index * n_points + point_index], tm, cm, d, B, A, n_batch, batch_index, n_points, point_index, xmin, xmax, nc);
        
    }
    return;
}

__global__ void kernel_integrate_closed_form_trace(
    const int n_points, const int n_batch, const float* x, const float* A, 
    const float t, const float xmin, const float xmax, const int nc, float* newpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    
    const int e = 3;

    if(point_index < n_points && batch_index < n_batch) {
        float result[e];
        integrate_closed_form_trace(result, x[batch_index * n_points + point_index], t, A, n_batch, batch_index, xmin, xmax, nc);
        for(int p = 0; p < e; p++){
            newpoints[batch_index*(n_points * e) + point_index*e + p] = result[p];
        }
    }
    return;
}

__global__ void kernel_derivative_closed_form_trace(
    const int n_points, const int n_batch, const int d,
    const float* newpoints, const float* x, const float* A, const float* B, 
    const float xmin, const float xmax, const int nc, double* gradpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;

    const int e = 3;

    if(point_index < n_points && batch_index < n_batch) {
        // float phi = newpoints[batch_index*(n_points * e) + point_index*e + 0];
        float tm = newpoints[batch_index*(n_points * e) + point_index*e + 1];
        int cm = newpoints[batch_index*(n_points * e) + point_index*e + 2];
        
        derivative_phi_theta(gradpoints, x[batch_index * n_points + point_index], tm, cm, d, B, A, n_batch, batch_index, n_points, point_index, xmin, xmax, nc);
    }
    return;
}



// INTERPOLATE
__device__ float clip(int num, int lower, int upper) {
    return max(lower, min(num, upper));
}
  
__device__ float interpolate_grid_forward(const float* x, const int& n_points, const int& batch_index, const int& point_index){
    
    float xc = x[batch_index*n_points + point_index]*(n_points - 1);
    int x0 = (int) std::floor(xc);
    int x1 = x0 + 1;
    x0 = clip(x0, 0, n_points-1);
    x1 = clip(x1, 0, n_points-1);
    float y0 = x[batch_index*n_points + x0];
    float y1 = x[batch_index*n_points + x1];
    float xd = (float) xc - x0;

    return y0 * (1 - xd) + y1 * xd;
}


__global__ void kernel_interpolate_grid_forward(
    const int n_points, const int n_batch, const float* x, float* y){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    if(point_index < n_points && batch_index < n_batch) {
        y[batch_index * n_points + point_index] = interpolate_grid_forward(x, n_points, batch_index, point_index);
    }
    return;
}

__device__ void interpolate_grid_backward(float* gradient, const float* g, const float* x, const int& n_points, const int& batch_index, const int& point_index){
    
    int pos = n_points*batch_index + point_index;
            
    float xc = x[pos]*(n_points - 1);
    int x0 = (int) std::floor(xc);
    int x1 = x0 + 1;
    x0 = clip(x0, 0, n_points-1);
    x1 = clip(x1, 0, n_points-1);
    float y0 = x[batch_index*n_points + x0];
    float y1 = x[batch_index*n_points + x1];
    float xd = (float) xc - x0;

    gradient[n_points*batch_index + x0] += (1-xd) * g[pos];
    gradient[n_points*batch_index + x1] += xd * g[pos];
    gradient[n_points*batch_index + point_index] += (n_points-1)*(y1-y0) * g[pos];
}


__global__ void kernel_interpolate_grid_backward(
    const int n_points, const int n_batch, const float* g, const float* x, float* gradient){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    if(point_index < n_points && batch_index < n_batch) {
        interpolate_grid_backward(gradient, g, x, n_points, batch_index, point_index);
    }
    return;
}


// GRADIENT SPACE

__device__ float derivative_thit_x(const float& x, const int& c, const float& t, const float* A, const int& n_batch, const int& batch_index){
    const float a = A[(2*c) * n_batch + batch_index];
    const float b = A[(2*c+1) * n_batch + batch_index];
    return  1.0 / (a*x + b);
}

__device__ float derivative_psi_x(const float& x, const int& c, const float& t, const float* A, const int& n_batch, const int& batch_index){
    const float a = A[(2*c) * n_batch + batch_index];
    // const float b = A[(2*c+1) * n_batch + batch_index];
    return  exp(t*a);
}

__device__ float derivative_psi_t(const float& x, const int& c, const float& t, const float* A, const int& n_batch, const int& batch_index){
    const float a = A[(2*c) * n_batch + batch_index];
    const float b = A[(2*c+1) * n_batch + batch_index];
    return  exp(t*a)*(a*x + b);
}

__device__ float derivative_phi_x(const float& xini, const float& tini, const float& tm, const int& cm, const float* A, const int& n_batch, const int& batch_index, const int& n_points, const int& point_index, const float& xmin, const float& xmax, const int& nc){
    
    const int cini = get_cell(xini, xmin, xmax, nc);
    float xm = xini;

    float dpsi_dx = 0.0;
    float dthit_dx = 0.0;
    if (cini == cm){
        dpsi_dx = derivative_psi_x(xini, cini, tini, A, n_batch, batch_index);
    }else{
        dthit_dx = derivative_thit_x(xini, cini, tini, A, n_batch, batch_index);
    }

    if (cini != cm){
        float xc;
        const int step = sign(cm - cini);
        for (int c = cini; step*c < cm*step; c += step){
            if (step == 1){
                xc = right_boundary(c, xmin, xmax, nc);
            }else if (step == -1){
                xc = left_boundary(c, xmin, xmax, nc);
            }
            xm = xc;
        }
    }

    float dpsi_dtime = derivative_psi_t(xm, cm, tm, A, n_batch, batch_index);
    float dphi_dx = dpsi_dx + dpsi_dtime * dthit_dx;
   
    return dphi_dx;
    
}

__global__ void kernel_derivative_space_closed_form(
    const int n_points, const int n_batch, 
    const float* x, const float* A, 
    const float t, const int xmin, const int xmax, const int nc, double* gradpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    
    const int e = 3;

    if(point_index < n_points && batch_index < n_batch){ 
        float result[e];
        integrate_closed_form_trace(result, x[batch_index * n_points + point_index], t, A, n_batch, batch_index, xmin, xmax, nc);
            
        // float phi = result[0];
        float tm = result[1];
        int cm = result[2];
        float dphi_dx = derivative_phi_x(x[batch_index * n_points + point_index], t, tm, cm, A, n_batch, batch_index, n_points, point_index, xmin, xmax, nc);
        
        gradpoints[batch_index * n_points + point_index] = dphi_dx;
    }
    return;
}

// GRADIENT SPACE DERIVATIVE THETA

__device__ float derivative_psi_x_theta(const float& x, const int& c, const float& t, const float* A, const int& k, const int& d, const float* B, const int& n_batch, const int& batch_index){
    const double a = A[(2*c) * n_batch + batch_index];
    // const double b = A[(2*c+1) * n_batch + batch_index];

    const double ak = B[(2*c)*d + k];
    // const double bk = B[(2*c+1)*d + k];

    return t * exp(t*a) * ak;
}

__device__ float derivative_thit_x_theta(const float& x, const int& c, const float& t, const float* A, const int& k, const int& d, const float* B, const int& n_batch, const int& batch_index){
    const double a = A[(2*c) * n_batch + batch_index];
    const double b = A[(2*c+1) * n_batch + batch_index];

    const double ak = B[(2*c)*d + k];
    const double bk = B[(2*c+1)*d + k];

    return - (x*ak + bk)/std::pow(a*x + b, 2.0);
}

__device__ float derivative_psi_t_theta(const float& dtm, const float& x, const int& c, const float& t, const float* A, const int& k, const int& d, const float* B, const int& n_batch, const int& batch_index){
    const double a = A[(2*c) * n_batch + batch_index];
    const double b = A[(2*c+1) * n_batch + batch_index];

    const double ak = B[(2*c)*d + k];
    const double bk = B[(2*c+1)*d + k];

    return exp(t*a) * ( a*(a*x+b)*dtm + ak*(t*(a*x+b) + x) + bk);
}

__device__ float derivative_thit_theta_alt(const float& x, const int& c, const float& xc, const int& k, const int& d, const float* B, const float* A, const int& n_batch, const int& batch_index){
    const double a = A[(2*c) * n_batch + batch_index];
    const double b = A[(2*c+1) * n_batch + batch_index];

    const double ak = B[(2*c)*d + k];
    const double bk = B[(2*c+1)*d + k];

    if (cmpf0(a)){
        const double tmp = (x-xc) / pow(b, 2.0);
        return -(tmp*bk);
    }
    else{
        const double tmp1 = log( (a*xc + b) / (a*x + b) )/pow(a, 2.0);
        const double tmp2 = (x - xc) / (a * (a*x + b) * (a*xc + b) );

        const double d1 = - ak * tmp1;
        const double d2 = ( bk*a - ak*b) * tmp2;
        return -(d1+d2);
    }
}

__device__ void derivative_phi_x_theta(double* gradpoints, const float& xini, const float& tini, const float& tm, const int& cm, const int& d, const float* B, const float* A, const int& n_batch, const int& batch_index, const int& n_points, const int& point_index, const float& xmin, const float& xmax, const int& nc){
    
    const int cini = get_cell(xini, xmin, xmax, nc);
    
    for(int k=0; k < d; k++){
        float xm = xini;
        float dthit_dtheta_cum = 0.0;
        if (cini != cm){
            float xc;
            const int step = sign(cm - cini);
            for (int c = cini; step*c < cm*step; c += step){
                if (step == 1){
                    xc = right_boundary(c, xmin, xmax, nc);
                }else if (step == -1){
                    xc = left_boundary(c, xmin, xmax, nc);
                }
                dthit_dtheta_cum += derivative_thit_theta_alt(xm, c, xc, k, d, B, A, n_batch, batch_index);
                xm = xc;
            } 
        }
    
        float dpsi_dtime = derivative_psi_t(xm, cm, tm, A, n_batch, batch_index);

        float dthit_dx = 0.0;
        float dpsi_dx_dtheta = 0.0;
        float dthit_dx_dtheta = 0.0;
        if (cini == cm){
            dpsi_dx_dtheta = derivative_psi_x_theta(xini, cini, tini, A, k, d, B, n_batch, batch_index);
        }else{
            dthit_dx = derivative_thit_x(xini, cini, tini, A, n_batch, batch_index);
            dthit_dx_dtheta = derivative_thit_x_theta(xini, cini, tini, A, k, d, B, n_batch, batch_index);
        }
        float dpsi_dtime_dtheta = derivative_psi_t_theta(dthit_dtheta_cum, xm, cm, tm, A, k, d, B, n_batch, batch_index);
        float dphi_dx_dtheta = dpsi_dx_dtheta + dpsi_dtime_dtheta * dthit_dx + dpsi_dtime * dthit_dx_dtheta;
        gradpoints[batch_index*(n_points * d) + point_index*d + k] = dphi_dx_dtheta;
    }
}



__global__ void kernel_derivative_space_closed_form_dtheta(
    const int n_points, const int n_batch, const int d,
    const float* x, const float* A, const float* B, const float t,
    const float xmin, const float xmax, const int nc, double* gradpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;

    const int e = 3;

    if(point_index < n_points && batch_index < n_batch) {
        float result[e];
        integrate_closed_form_trace(result, x[batch_index * n_points + point_index], t, A, n_batch, batch_index, xmin, xmax, nc);
            
        // float phi = result[0];
        float tm = result[1];
        int cm = result[2];
        
        derivative_phi_x_theta(gradpoints, x[batch_index * n_points + point_index], t, tm, cm, d, B, A, n_batch, batch_index, n_points, point_index, xmin, xmax, nc);
    }
    return;
}




// GRADIENT SPACE DERIVATIVE X

__device__ float derivative_thit_x_x(const float& x, const int& c, const float& t, const float* A, const int& n_batch, const int& batch_index){
    const float a = A[(2*c) * n_batch + batch_index];
    const float b = A[(2*c+1) * n_batch + batch_index];

    return - a / std::pow(a*x + b, 2.0);
}

__device__ float derivative_psi_t_x(const float& x, const int& c, const float& t, const float* A, const int& n_batch, const int& batch_index){
    const float a = A[(2*c) * n_batch + batch_index];
    // const float b = A[(2*c+1) * n_batch + batch_index];

    return a * exp(t*a);
}

__device__ float derivative_phi_x_x(const float& xini, const float& tini, const float& tm, const int& cm, const float* A, const int& n_batch, const int& batch_index, const int& n_points, const int& point_index, const float& xmin, const float& xmax, const int& nc){
    const int cini = get_cell(xini, xmin, xmax, nc);
    float xm = xini;

    float dthit_dx = 0.0;
    float dthit_dx_dx = 0.0;
    if (cini != cm){
        dthit_dx = derivative_thit_x(xini, cini, tini, A, n_batch, batch_index);
        dthit_dx_dx = derivative_thit_x_x(xini, cini, tini, A, n_batch, batch_index);
    }


    if (cini != cm){
        float xc;
        const int step = sign(cm - cini);
        for (int c = cini; step*c < cm*step; c += step){
            if (step == 1){
                xc = right_boundary(c, xmin, xmax, nc);
            }else if (step == -1){
                xc = left_boundary(c, xmin, xmax, nc);
            }
            xm = xc;
        } 
    }

    float dpsi_dtime = derivative_psi_t(xm, cm, tm, A, n_batch, batch_index);
    float dpsi_dtime_dx = derivative_psi_t_x(xm, cm, tm, A, n_batch, batch_index);
    float dphi_dx = dpsi_dtime_dx * dthit_dx + dpsi_dtime * dthit_dx_dx;
    return dphi_dx;
}

__global__ void kernel_derivative_space_closed_form_dx(
    const int n_points, const int n_batch, 
    const float* x, const float* A, 
    const float t, const int xmin, const int xmax, const int nc, double* gradpoints){

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;
    
    const int e = 3;

    if(point_index < n_points && batch_index < n_batch){ 
        float result[e];
        integrate_closed_form_trace(result, x[batch_index * n_points + point_index], t, A, n_batch, batch_index, xmin, xmax, nc);
            
        // float phi = result[0];
        float tm = result[1];
        int cm = result[2];
        float dphi_dx_dx = derivative_phi_x_x(x[batch_index * n_points + point_index], t, tm, cm, A, n_batch, batch_index, n_points, point_index, xmin, xmax, nc);
        
        gradpoints[batch_index * n_points + point_index] = dphi_dx_dx;
    }
    return;
}